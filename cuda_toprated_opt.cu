#include "hip/hip_runtime.h"
// Full CUDA Bitonic Sort with Shared Memory and Optimizations
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <sstream>
#include <map>
#include <utility>
#include <algorithm>
#include <iomanip>
#include <stdexcept>
#include <numeric>
#include <cmath>
#include <chrono>

#include "rapidjson/document.h"
#include "rapidjson/error/en.h"
#include <hip/hip_runtime.h>

#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Failure in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    }

using namespace std;

__global__ void performBitonicStageShared(float *values, int *indices, int stage, int pass, int count) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    extern __shared__ char sharedMemory[];
    float *sharedVals = (float *)sharedMemory;
    int *sharedIndices = (int *)&sharedVals[blockDim.x];

    if (idx < count) {
        sharedVals[tid] = values[idx];
        sharedIndices[tid] = indices[idx];
    }

    __syncthreads();

    int pairDistance = 1 << (stage - pass);
    int blockSize = 1 << stage;
    int partner = idx ^ pairDistance;
    bool ascending = ((idx & blockSize) == 0);

    if (partner < count && tid < blockDim.x) {
        int localPartner = partner - blockIdx.x * blockDim.x;
        if (localPartner >= 0 && localPartner < blockDim.x) {
            float val1 = sharedVals[tid];
            float val2 = sharedVals[localPartner];
            int ind1 = sharedIndices[tid];
            int ind2 = sharedIndices[localPartner];

            bool doSwap = (ascending && val1 > val2) || (!ascending && val1 < val2);

            if (doSwap) {
                sharedVals[tid] = val2;
                sharedVals[localPartner] = val1;
                sharedIndices[tid] = ind2;
                sharedIndices[localPartner] = ind1;
            }
        }
    }

    __syncthreads();

    if (idx < count) {
        values[idx] = sharedVals[tid];
        indices[idx] = sharedIndices[tid];
    }
}

int calculateNextPower2(int x) {
    if (x <= 0) return 1;
    int p = 1;
    while (p < x) p *= 2;
    return p;
}

void computeAverages(const map<string, pair<double, int>> &aggregates,
                     vector<float> &ratings,
                     vector<string> &products,
                     vector<int> &original_indices) {
    ratings.reserve(aggregates.size());
    products.reserve(aggregates.size());
    original_indices.reserve(aggregates.size());
    int index = 0;
    for (const auto &entry : aggregates) {
        if (entry.second.second > 0) {
            ratings.push_back(static_cast<float>(entry.second.first / entry.second.second));
            products.push_back(entry.first);
            original_indices.push_back(index++);
        }
    }
}

int main(int argc, char **argv) {
    auto start_time = chrono::high_resolution_clock::now();

    if (argc != 2) {
        cerr << "Provide JSON file path as argument.\n";
        return 1;
    }

    ifstream input(argv[1]);
    if (!input.is_open()) {
        cerr << "Unable to read file: " << argv[1] << endl;
        return 1;
    }

    cout << "Loading and aggregating reviewsd..." << endl;
    map<string, pair<double, int>> review_map;
    string row;
    long long lines = 0, valid_lines = 0, bad_lines = 0;
    rapidjson::Document doc;

    while (getline(input, row)) {
        lines++;
        doc.Parse(row.c_str());
        if (doc.HasParseError()) { bad_lines++; continue; }

        if (doc.IsObject() && doc.HasMember("asin") && doc["asin"].IsString()
            && doc.HasMember("overall") && doc["overall"].IsNumber()) {
            try {
                string id = doc["asin"].GetString();
                float rating = doc["overall"].GetFloat();
                review_map[id].first += rating;
                review_map[id].second++;
                valid_lines++;
            } catch (const exception &e) {
                bad_lines++;
            }
        } else bad_lines++;
    }
    input.close();

    cout << "Reviews loaded: " << valid_lines << " for " << review_map.size() << " items." << endl;
    if (bad_lines > 0) cout << "Ignored " << bad_lines << " malformed entries." << endl;
    if (review_map.empty()) return 1;

    vector<float> averages;
    vector<string> asin_list;
    vector<int> indices;
    computeAverages(review_map, averages, asin_list, indices);
    int total_items = averages.size();
    if (total_items == 0) return 1;

    cout << "Sorting using GPU..." << endl;

    hipEvent_t t_start, t_end;
    CUDA_CHECK(hipEventCreate(&t_start));
    CUDA_CHECK(hipEventCreate(&t_end));
    float h2d_time = 0, exec_time = 0, d2h_time = 0;

    int padded = calculateNextPower2(total_items);
    vector<float> padded_values(padded, -1.0f);
    vector<int> padded_ids(padded, -1);
    copy(averages.begin(), averages.end(), padded_values.begin());
    copy(indices.begin(), indices.end(), padded_ids.begin());

    float *d_vals;
    int *d_inds;
    CUDA_CHECK(hipEventRecord(t_start));
    CUDA_CHECK(hipMalloc(&d_vals, padded * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_inds, padded * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_vals, padded_values.data(), padded * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_inds, padded_ids.data(), padded * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipEventRecord(t_end));
    CUDA_CHECK(hipEventSynchronize(t_end));
    CUDA_CHECK(hipEventElapsedTime(&h2d_time, t_start, t_end));

    int steps = static_cast<int>(log2(padded));
    int threads = 256;
    int blocks = (padded + threads - 1) / threads;
    int sharedMemSize = threads * (sizeof(float) + sizeof(int));

    CUDA_CHECK(hipEventRecord(t_start));
    for (int stage = 0; stage < steps; ++stage) {
        for (int pass = 0; pass <= stage; ++pass) {
            performBitonicStageShared<<<blocks, threads, sharedMemSize>>>(d_vals, d_inds, stage + 1, pass + 1, padded);
        }
    }
    CUDA_CHECK(hipEventRecord(t_end));
    CUDA_CHECK(hipEventSynchronize(t_end));
    CUDA_CHECK(hipEventElapsedTime(&exec_time, t_start, t_end));

    vector<int> sorted_indices(padded);
    CUDA_CHECK(hipEventRecord(t_start));
    CUDA_CHECK(hipMemcpy(sorted_indices.data(), d_inds, padded * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_vals));
    CUDA_CHECK(hipFree(d_inds));
    CUDA_CHECK(hipEventRecord(t_end));
    CUDA_CHECK(hipEventSynchronize(t_end));
    CUDA_CHECK(hipEventElapsedTime(&d2h_time, t_start, t_end));

    CUDA_CHECK(hipEventDestroy(t_start));
    CUDA_CHECK(hipEventDestroy(t_end));

    cout << "GPU sort complete.\n\n--- Top 10 Products ---\n";
    int top_n = min(total_items, 10), shown = 0;
    cout << fixed << setprecision(4);
    for (int i = 0; i < padded && shown < top_n; ++i) {
        int idx = sorted_indices[i];
        if (idx >= 0 && idx < total_items) {
            cout << shown + 1 << ". Product ID: " << asin_list[idx]
                 << " | Rating: " << averages[idx] << endl;
            shown++;
        }
    }

    auto end_time = chrono::high_resolution_clock::now();
    chrono::duration<double, milli> full_duration = end_time - start_time;

    cout << "\n--- Performance Stats ---\n";
    cout << fixed << setprecision(3);
    cout << "Memory Transfer to GPU:       " << h2d_time << " ms\n";
    cout << "GPU Bitonic Sort Time:        " << exec_time << " ms\n";
    cout << "Transfer from GPU & Cleanup:  " << d2h_time << " ms\n";
    cout << "Total Runtime:                " << full_duration.count() << " ms\n";

    return 0;
}
